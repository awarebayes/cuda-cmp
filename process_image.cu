#include <hip/hip_runtime.h>

extern "C" __global__ void process_image(
    const unsigned char* input,
    unsigned char* output,
    int width,
    int height,
    int batch_size
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int img_idx = blockIdx.z;

    if (x < width && y < height && img_idx < batch_size) {
        int pixel_idx = img_idx * width * height * 3 + (y * width + x) * 3;

        // Copy the pixel values from input to output
        output[pixel_idx] = input[pixel_idx];       // Red
        output[pixel_idx + 1] = input[pixel_idx + 1]; // Green
        output[pixel_idx + 2] = input[pixel_idx + 2]; // Blue
    }
}
